#include "hip/hip_runtime.h"
/*
	Linux build command
	/usr/local/cuda/bin/nvcc -o xva xva.cu -std=c++11 -O3
*/

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/iterator/zip_iterator.h>
#include <algorithm>
#include <iostream>
#include <chrono> 
#include <thread>  
#include <mutex>
#include <atomic>

typedef double DataType;
typedef thrust::tuple<DataType,DataType,DataType,DataType,DataType> DataTypeTuple5;
typedef std::chrono::time_point<std::chrono::high_resolution_clock> HRTime;

struct transorm_func
{ 
  __host__ __device__ DataType operator()(const DataTypeTuple5 &a) const 
  {
	  const DataType& cp = thrust::get<0>(a);
	  const DataType& spot_ccy1 = thrust::get<1>(a);
	  const DataType& param_ccy1 = thrust::get<2>(a);
	  const DataType& spot_ccy2 = thrust::get<3>(a);
	  const DataType& param_ccy2 = thrust::get<4>(a);

	  DataType res = cp + spot_ccy1 * param_ccy1 - spot_ccy2 - param_ccy2;
	  return fmin(fmax(0, res), res * 2.0);
  }
};

template <typename T1, typename T2>
void host_to_device(T1& hostVec, T2& devVec) 
{
	thrust::copy(hostVec.begin(), hostVec.end(), devVec.begin());
}

template <typename T1, typename T2>
void device_to_host(T1& hostVec, T2& devVec) 
{
	thrust::copy(devVec.begin(), devVec.end(), hostVec.begin());
}

template <typename T1, typename T2>
void host_to_device(hipStream_t& stream, T1& hostVec, T2& devVec, int elementSize) 
{
	hipMemcpyAsync(thrust::raw_pointer_cast(devVec.data()), thrust::raw_pointer_cast(hostVec.data()), hostVec.size()*elementSize, hipMemcpyHostToDevice, stream);
}

template <typename T1, typename T2>
void device_to_host(hipStream_t& stream, T1& hostVec, T2& devVec, int elementSize) 
{
	hipMemcpyAsync(thrust::raw_pointer_cast(hostVec.data()), thrust::raw_pointer_cast(devVec.data()), devVec.size()*elementSize, hipMemcpyDeviceToHost, stream);
}

int main(int argc, const char* argv[])
{
	for (int threadsCount = 1; threadsCount < 65; threadsCount++)
	{
		int datesCount = 180;
		int simulationsCount = 10000;
		int size = datesCount * simulationsCount;
		
		thrust::host_vector<int> h_keys(size);
		for (int i = 0; i < size; i++) 
		{
			h_keys[i] = i % datesCount;
		}
		std::sort(h_keys.begin(), h_keys.end());

		thrust::host_vector<int> h_keys_res(datesCount);
		thrust::host_vector<DataType> h_cp(size);
		
		thrust::host_vector<DataType> h_spot_ccy1(size);
		thrust::host_vector<DataType> h_param_ccy1(size);
		thrust::host_vector<DataType> h_spot_ccy2(size);
		thrust::host_vector<DataType> h_param_ccy2(size);
		
		std::generate(h_cp.begin(), h_cp.end(), rand); 
		std::generate(h_spot_ccy1.begin(), h_spot_ccy1.end(), rand); 
		std::generate(h_param_ccy1.begin(), h_param_ccy1.end(), rand); 
		std::generate(h_spot_ccy2.begin(), h_spot_ccy2.end(), rand); 
		std::generate(h_param_ccy2.begin(), h_param_ccy2.end(), rand); 
		std::generate(h_keys_res.begin(), h_keys_res.end(), rand); 
		
		std::vector<int> executionTimes;
		for (int r = 0; r < 20; r ++) 
		{
			thrust::device_vector<DataType> d_keys(size); 
			thrust::device_vector<DataType> d_cp(size);
			thrust::device_vector<DataType> d_spot_ccy1(size);
			thrust::device_vector<DataType> d_param_ccy1(size);
			thrust::device_vector<DataType> d_spot_ccy2(size);
			thrust::device_vector<DataType> d_param_ccy2(size);
			
			HRTime dataTransferStart = std::chrono::high_resolution_clock::now(); 
			
			host_to_device(h_keys, d_keys);
			host_to_device(h_cp, d_cp);
			host_to_device(h_spot_ccy1, d_spot_ccy1);
			host_to_device(h_param_ccy1, d_param_ccy1);
			host_to_device(h_spot_ccy2, d_spot_ccy2);
			host_to_device(h_param_ccy2, d_param_ccy2);
			
			std::vector<std::thread> threads;
			std::mutex lock;
		
			for (int threadId = 0; threadId < threadsCount; threadId++) 
			{
				std::atomic_int latch(threadsCount);
				threads.push_back(std::thread([&, threadId]()
				{
					hipStream_t stream;
					hipStreamCreate(&stream);
					
					thrust::host_vector<DataType> h_epe(size);
					thrust::host_vector<DataType> h_date_epe(datesCount);
					
					thrust::device_vector<DataType> d_epe(size);
					thrust::device_vector<DataType> d_date_epe(datesCount);
					thrust::device_vector<int> d_keys_output(datesCount);
					
					hipStreamSynchronize(stream);
					
					HRTime gpuAlgorithmStart = std::chrono::high_resolution_clock::now();
					
					host_to_device(stream, h_keys_res, d_keys_output, sizeof(int)); // simulate sending parameters
					
					hipStreamSynchronize(stream);
					
					auto begin = thrust::make_zip_iterator(thrust::make_tuple(d_cp.begin(), d_spot_ccy1.begin(), d_param_ccy1.begin(), d_spot_ccy2.begin(), d_param_ccy2.begin()));
					auto end = thrust::make_zip_iterator(thrust::make_tuple(d_cp.end(), d_spot_ccy1.end(), d_param_ccy1.end(), d_spot_ccy2.end(), d_param_ccy2.end()));
					thrust::transform(thrust::cuda::par.on(stream),
									  begin, 
									  end, 
									  d_epe.begin(), 
									  transorm_func()); 

					thrust::reduce_by_key(thrust::cuda::par.on(stream), d_keys.begin(), d_keys.end(), d_epe.begin(), d_keys_output.begin(), d_date_epe.begin());

					/* PFE Sorting
					auto keysBegin = thrust::make_zip_iterator(thrust::make_tuple(d_keys.begin(), d_epe.begin()));
					auto keysEnd = thrust::make_zip_iterator(thrust::make_tuple(d_keys.end(), d_epe.end()));
					
					thrust::sort_by_key(thrust::cuda::par.on(stream),
										keysBegin,
										keysEnd,
										d_epe.begin(),
										thrust::less<thrust::tuple<int,DataType>>());
					*/   
					HRTime hostTransferStart = std::chrono::high_resolution_clock::now();
					
					device_to_host(stream, h_date_epe, d_date_epe, sizeof(DataType));
					device_to_host(stream, h_keys_res, d_keys_output, sizeof(int));
					
					hipStreamSynchronize(stream);
					
					HRTime hostTransferEnd = std::chrono::high_resolution_clock::now();
					
					hipStreamDestroy(stream);
					
					int64_t totalTime = std::chrono::duration_cast<std::chrono::microseconds>(hostTransferEnd - dataTransferStart).count();
					int64_t executionTime = std::chrono::duration_cast<std::chrono::microseconds>(hostTransferEnd - gpuAlgorithmStart).count();
					
					std::lock_guard<std::mutex> guard(lock);
					executionTimes.push_back(executionTime);
				}));
			}
			
			for (auto& thread : threads)
			{
				thread.join();
			}
		}
		int min = *std::min_element(executionTimes.begin(), executionTimes.end());
		int max = *std::max_element(executionTimes.begin(), executionTimes.end());
		double avg = std::accumulate(executionTimes.begin(), executionTimes.end(), 0.0) / executionTimes.size();
		std::sort(executionTimes.begin(), executionTimes.end());
		int median = executionTimes[executionTimes.size() / 2];
		int perc90 = executionTimes[(int)((double)executionTimes.size() * 0.9)];
		printf("Threads count:\t%d Execution time min:\t%d\tmax:\t%d\tavg:\t%.f\tmedian:\t%d\t90%:\t%d\tmicroseconds\n", threadsCount, min, max, avg, median, perc90);
	}
	return 0;
}

