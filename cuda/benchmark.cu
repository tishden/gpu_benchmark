#include "hip/hip_runtime.h"
/*
    /usr/local/cuda/bin/nvcc -o benchmark benchmark.cu -std=c++11 -O3
*/

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <iostream>
#include <chrono>

#include "VariadicTable.h"

typedef double DataType;
typedef std::chrono::time_point<std::chrono::high_resolution_clock> HRTime;

enum Device {
    gpu,
    cpu,
    last
};

enum Algorithm {
    sort,
    sum,
    count_if,
    find_max,
    transform,
    last_alg
};

template <typename T>
struct greater_than_five
{
  __host__ __device__ bool operator()(const T &x) const {return x > 5;}
};

template <typename T>
struct transorm_func
{ 
  __host__ __device__ T operator()(const T &x) const {return x * x + x / 3;}
};

int main(int argc, const char* argv[])
{
    VariadicTable<int, std::string, double, double, double, double, double,  double, double> vt({"Elements", "Algorithm", "HtoD Time, ms", " GPU Execution, ms", "DtoH, ms", "GPU Total Time, ms", "CPU Time, ms", "Diff", "Execution + DtoH Diff"});
    int sizes[] = {1, 1, 100, 1000, 10000, 100000, 1000000, 10000000, 100000000, 500000000};
    
    for (int iAlgorithm = Algorithm::sort; iAlgorithm != Algorithm::last_alg; iAlgorithm++)
    {   
        for (const int& size : sizes)
        {
            Algorithm algorithm = (Algorithm)iAlgorithm;
            std::string algorithmStr;
             
            HRTime dataTransferStart;        
            HRTime hostTransferStart;
            HRTime hostTransferEnd;

            HRTime gpuAlgorithmStart;
            
            HRTime cpuAlgorithmStart;
            HRTime cpuAlgorithmEnd;
            
            thrust::host_vector<DataType> h_vec(size);
            thrust::host_vector<DataType> h_result_vec(size);
     
            // generate random numbers serially
            std::generate(h_vec.begin(), h_vec.end(), rand); 
            
            for (int iDevice = Device::gpu; iDevice != Device::last; iDevice++)
            {   
                Device device = (Device)iDevice;
                if (device == gpu)
                {
                    // transfer data to the device
                    dataTransferStart = std::chrono::high_resolution_clock::now();
                    thrust::device_vector<DataType> d_vec = h_vec;

                    gpuAlgorithmStart = std::chrono::high_resolution_clock::now();

                    // initial value of the reduction
                    DataType init = 0; 
                    DataType result = 0;
                    switch (algorithm)
                    {
                        case (sort):
                        {
                            algorithmStr = "sort";
                            thrust::sort(d_vec.begin(), d_vec.end());
                            // transfer data back to host
                            hostTransferStart = std::chrono::high_resolution_clock::now();
                            thrust::copy(d_vec.begin(), d_vec.end(), h_result_vec.begin());
                            break;
                        }
                        case (sum):
                        {
                           algorithmStr = "sum";
                            // binary operation used to reduce values
                            thrust::plus<DataType> binary_op;
                            // compute sum on the device
                            result = thrust::reduce(d_vec.begin(), d_vec.end(), init, binary_op);
                            hostTransferStart = std::chrono::high_resolution_clock::now();
                            break;
                        }
                        case (count_if):
                        {
                            algorithmStr = "count_if";
                            result = thrust::count_if(d_vec.begin(), d_vec.end(), greater_than_five<DataType>());                    
                            hostTransferStart = std::chrono::high_resolution_clock::now();
                            break;
                        }
                        case (find_max):
                        {
                            algorithmStr = "max";
                            thrust::device_vector<DataType>::iterator iter = thrust::max_element(d_vec.begin(), d_vec.end());
                            result = *iter;
                            hostTransferStart = std::chrono::high_resolution_clock::now();
                            break;
                        }
                        case (transform):
                        {
                            algorithmStr = "transform";
                            thrust::transform(d_vec.begin(),d_vec.end(),d_vec.begin(),transorm_func<DataType>());
                            // transfer data back to host
                            hostTransferStart = std::chrono::high_resolution_clock::now();
                            thrust::copy(d_vec.begin(), d_vec.end(), h_result_vec.begin());
                            break;
                        }
                    }
                    hostTransferEnd = std::chrono::high_resolution_clock::now();
                    std::cout << result << std::endl;
                }
                else if (device == cpu)
                {
                    DataType result = 0;
                    cpuAlgorithmStart = std::chrono::high_resolution_clock::now(); 
                    switch (algorithm)
                    {
                        case (sort):
                        {
                            algorithmStr = "sort";
                            std::sort(h_vec.begin(), h_vec.end());
                            break;
                        }
                        case (sum):
                        {
                            algorithmStr = "sum";
                            result = std::accumulate(h_vec.begin(), h_vec.end(), 0);
                            break;
                        }
                        case (count_if):
                        {
                            algorithmStr = "count_if";
                            result = std::count_if(h_vec.begin(), h_vec.end(), [](DataType i){return i > 5;});
                            break;    
                        }                        
                        case (find_max):
                        {
                            algorithmStr = "max";
                            thrust::host_vector<DataType>::iterator resultIt; 
                            resultIt = std::max_element(h_vec.begin(), h_vec.end());
                            result = *resultIt;
                            break;
                        }
                        case (transform):
                        {
                            algorithmStr = "transform";
                            std::transform(h_vec.begin(), h_vec.end(), h_vec.begin(), transorm_func<DataType>());
                            break;
                        }
                    }
                    cpuAlgorithmEnd = std::chrono::high_resolution_clock::now();
                    std::cout << result << std::endl;
                }
            }
            int64_t hToD = std::chrono::duration_cast<std::chrono::microseconds>(gpuAlgorithmStart - dataTransferStart).count();
            int64_t executionTime = std::chrono::duration_cast<std::chrono::microseconds>(hostTransferStart - gpuAlgorithmStart).count();
            int64_t dToH = std::chrono::duration_cast<std::chrono::microseconds>(hostTransferEnd - hostTransferStart).count();
            int64_t totalTime = std::chrono::duration_cast<std::chrono::microseconds>(hostTransferEnd - dataTransferStart).count();
            int64_t cpuTime = std::chrono::duration_cast<std::chrono::microseconds>(cpuAlgorithmEnd - cpuAlgorithmStart).count();
            double diff = cpuTime / totalTime;       
            double diff2 = cpuTime / (executionTime + dToH);       
            vt.addRow(std::make_tuple(size, algorithmStr, hToD / 1000.0, executionTime / 1000.0, dToH / 1000.0, totalTime / 1000.0, cpuTime / 1000.0, diff, diff2));
        }
    }    
    vt.print(std::cout);
    
    return 0;
}